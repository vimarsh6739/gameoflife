#include "hip/hip_runtime.h"
#include "CPU_gol.h"
#include <random>

__global__ void updateStateColorKernel(int* curr_state_dev, int rows, int columns, int* next_state_dev, float* state_color_dev)
{
	//Finding the unique id of this thread
	int tid = blockIdx.x*blockDim.x+threadIdx.x;

	if(tid < rows * columns)
	{
		curr_state_dev[tid] = next_state_dev[tid];
		
		//Can be changed later for colored update keeping track of previous state
		state_color_dev[3*tid] = (float)next_state_dev[tid];
		state_color_dev[3*tid+1] = (float)next_state_dev[tid];
		state_color_dev[3*tid+2] = (float)next_state_dev[tid];
	}
}

__global__ void findNextStateKernel(int* curr_state_dev, int rows, int columns, int* next_state_dev)
{
	//Finding the unique id of this thread
	int tid=blockIdx.x*blockDim.x+threadIdx.x;

	//performing the operations only for the threads corresponding to a cell in the grid
	if(tid < rows*columns)
	{
		//Get the cyclic count of neighbours around the current cell
		int nbrCnt = 0;
		int top,bottom,mid_r,left,mid_c,right;
		
		//Current row and column
		int i = tid / columns;
		int j = tid % columns;
		
		//Defining variables for the grid surrounding (i,j)
		top=((i-1+rows)%rows)*columns;
		bottom=((i+1)%rows)*columns;
		mid_r = i*columns;
		mid_c =j;
		left = (j-1+columns)%columns;
		right = (j+1)%columns;

		//Counting the number of neighbours around (i,j)
		nbrCnt =  curr_state_dev[top + left] 
		   		+ curr_state_dev[top + mid_c] 
		   		+ curr_state_dev[top + right]
		   		+ curr_state_dev[mid_r + left]  
		   		+ curr_state_dev[mid_r + right]
		   		+ curr_state_dev[bottom + left] 
		   		+ curr_state_dev[bottom + mid_c] 
		   		+ curr_state_dev[bottom + right];
		
		if(nbrCnt == 3 || ((nbrCnt==2)&&(curr_state_dev[tid]==1)))
		{
			next_state_dev[tid] = 1;
		}
		else
		{
			next_state_dev[tid] = 0;
		}
	}
}

CPU_gol::CPU_gol()
{
	//Empty default constructor
	// :)
}

//Allot memory for state bitmap
CPU_gol::CPU_gol(int rows, int columns)
{
	srand(time(NULL));

	this->rows = rows;
	this->columns = columns;
	N = rows*columns;

	this->curr_state = (int*)calloc(N,sizeof(int));
	this->next_state = (int*)calloc(N,sizeof(int));
	this->state_color = (float*)calloc(3*N,sizeof(float));

	this->updateIter = 0;

	this->isGpu = false;
}

//Allot memory for state bitmap
CPU_gol::CPU_gol(int rows, int columns, bool isGpu)
{
	srand(time(NULL));

	this->rows = rows;
	this->columns = columns;
	N = rows*columns;

	this->curr_state = (int*)calloc(N,sizeof(int));
	this->next_state = (int*)calloc(N,sizeof(int));
	this->state_color = (float*)calloc(3*N,sizeof(float));

	this->updateIter = 0;

	this->isGpu = isGpu;

	if(isGpu)
	{
		//Allocate memory in GPU for all cuda arrays
		hipMalloc(&curr_state_dev, N*sizeof(int));
		hipMalloc(&next_state_dev, N*sizeof(int));
		hipMalloc(&state_color_dev, 3*N*sizeof(float));

		//Set all arrays to 0
		hipMemset(curr_state_dev, 0, N*sizeof(int));
		hipMemset(next_state_dev, 0, N*sizeof(int));
		hipMemset(state_color_dev, 0, 3*N*sizeof(float));
	}
}

void CPU_gol::randInit()
{
	int doa = 0;
	int currPos = 0;
	for(int i=0; i<rows; ++i)
	{
		for(int j=0;j<columns;++j)
		{
			currPos = i*columns + j;
			doa = rand() % 2;
			curr_state[currPos] = doa;
			state_color[3*currPos] = (float)doa;
			state_color[3*currPos+1] = (float)doa;
			state_color[3*currPos+2] = (float)doa;
		}
	}

	//Update the next state for the current state in CPU or corresponding GPU
	if(isGpu)
	{
		//Copy current state and colors
		hipMemcpy(curr_state_dev, curr_stat, N*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(state_color_dev, state_color, 3*N*sizeof(float), hipMemcpyHostToDevice);

		int nblocks = ceil((m*n)/1024.0);;

		//Call a kernel to compute the next state value
		findNextStateKernel<<<nblocks,1024>>>(curr_state_dev,rows,columns,next_state_dev);

		//Store a copy of the next state in CPU
		hipMemcpy(next_state, next_state_dev, N*sizeof(int), hipMemcpyDeviceToHost);
	}
	else
	{
		findNextState();
	}
}

int CPU_gol::getNeighbourCount(int top, int mid_r, int bottom, int left, int mid_c, int right)
{
	return   curr_state[top + left] 
		   + curr_state[top + mid_c] 
		   + curr_state[top + right]
		   + curr_state[mid_r + left]  
		   + curr_state[mid_r + right]
		   + curr_state[bottom + left] 
		   + curr_state[bottom + mid_c] 
		   + curr_state[bottom + right];
}

void CPU_gol::findNextState()
{
	int currPos =0;
	int nbrCnt = 0;
	int top,bottom,mid_r,left,mid_c,right;
	for(int i=0;i<rows;++i)
	{
		top=((i-1+rows)%rows)*columns;
		bottom=((i+1)%rows)*columns;
		mid_r = i*columns;

		for(int j=0;j<columns;++j)
		{
			currPos = mid_r + j;
			mid_c =j;
			left = (j-1+columns)%columns;
			right = (j+1)%columns;
			nbrCnt = getNeighbourCount(top,mid_r,bottom,left,mid_c,right);
			if(nbrCnt == 3 || ((nbrCnt==2)&&(curr_state[currPos]==1)))
			{
				next_state[currPos] = 1;
			}
			else
			{
				next_state[currPos] = 0;
			}
		}
	}
}

void CPU_gol::updateState()
{
	updateIter++;
	
	//Update next_state
	if(isGpu)
	{
		int nblocks = ceil((m*n)/1024.0);;
		//Update colurs for next state in GPU
		updateStateColorKernel<<<nblocks,1024>>>(curr_state_dev,rows,columns,next_state_dev);
		
		//Call a kernel to compute the next state value
		findNextStateKernel<<<nblocks,1024>>>(curr_state_dev,rows,columns,next_state_dev);

	}
	else
	{	
		//Update colours for current state in CPU
		int currPos =0;
		for(int i=0;i<rows;++i)
		{
			for(int j=0;j<columns;++j)
			{
				currPos= i*columns + j;
				curr_state[currPos] = next_state[currPos];

				//Can be changed later for colored update keeping track of previous state
				state_color[3*currPos] = (float)next_state[currPos];
				state_color[3*currPos+1] = (float)next_state[currPos];
				state_color[3*currPos+2] = (float)next_state[currPos];
			}
		}

		//Find the next state value
		findNextState();
	}
}

bool CPU_gol::isAlive(int i, int j) 
{
	int currPos = i*columns + j;
	return curr_state[currPos]==1 ;
}

float* CPU_gol::getStateColours()
{
	return state_color;
}

void CPU_gol::printCells()
{
	int currPos = 0;
	for(int i=0;i < rows; ++i)
	{
		for(int j = 0; j < columns ;++j)
		{
			currPos = i*columns + j;
			std::cout<<curr_state[currPos]<<" ";
		}
		std::cout<<"\n";
	}
}

void CPU_gol::printColors()
{
	int currPos=0;
	for(int i=0;i<rows;++i)
	{
		for(int j=0;j<columns;++j)
		{
			currPos = i*columns + j;
			std::cout<<"("<<state_color[3*currPos]<<","<<state_color[3*currPos+1]<<","<<state_color[3*currPos+2]<<") ";
		}
		std::cout<<"\n";
	}
}